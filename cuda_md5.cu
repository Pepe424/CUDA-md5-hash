#include "hip/hip_runtime.h"
#include "cuda_md5.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define THREADS 512
#define GPU 4

__global__ void kernel_mult(int grid, unsigned long long block_length, int gpu, int len)
{
	uint8_t result[16];
	char msg[12] = {0};
	
	unsigned long long iter = (THREADS * blockIdx.x + threadIdx.x) + (grid * block_length) + (((block_length + gpu - 1)/ 4) * gpu); // 1. číslo iterace| 2. posun o grid| 3. posun o čtvrtinu(rozdělení mezi 4 gpu)

	for (size_t i = 0; i < len; i++)
	{
		msg[i] = iter % 26 + 97;
		iter = iter / 26;
	}

	cuda_md5((uint8_t *)msg, len, result);
}

void run_mult(int l)
{
	unsigned long long total = pow(26, l);
	unsigned long blocks = (total + THREADS - 1) / THREADS;
	unsigned long _blocks = blocks;
	printf("String length [%d] - Threads [%d] - Blocks limitations [%.0f]\n", l, THREADS, (pow(2, 31) - 1));
	printf("%lu <- Teoretical Blocks\n", blocks);

	int divide = (blocks + (pow(2, 31) - 1) - 1) / (pow(2, 31) - 1);
	blocks = (blocks + divide - 1) / divide;

	printf("%lu <- Actual Blocks\n", blocks * divide);
	printf("%lu <- Waster Blocks\n", blocks * divide - _blocks);
	printf("%llu <- Teoretical Operations\n", total);
	printf("%lu <- Actual Operations\n", blocks * THREADS * divide);
	printf("%llu <- Waste Operatins\n", blocks * THREADS * divide - total);
	printf("%d <- Divisor\n", divide);

	for (size_t i = 0; i < divide; i++)
	{
		for (size_t j = 0; j < GPU; j++)
		{
			hipSetDevice(j);
			kernel_mult<<<(blocks + GPU - 1)/GPU, THREADS>>>(i, blocks, j, l);
		}
		hipDeviceSynchronize();
	}
}