#include "hip/hip_runtime.h"
#include "cuda_md5.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define THREADS 512

__global__ void kernel_mult(int grid, unsigned long long block_length, int gpu, int len)
{
	uint8_t result[16];
	password pwd;

	pwd.length = len;
	unsigned long long iter = (BLOCK_SIZE * blockIdx.x + threadIdx.x) + (grid * block_length) + ((block_length/4) * gpu); //1. číslo iterace| 2. posun o grid| 3. posun o čtvrtinu(rozdělení mezi 4 gpu)
	for (size_t i = 0; i < len; i++)
	{
		pwd.word[i] = iter % 26 + 97;
		iter = iter / 26;
	}
	cuda_md5(&pwd, result);
}

void run_mult(int l)
{
	unsigned long long total = pow(26, l);
	unsigned long blocks = (total + THREADS - 1) / THREADS;
	printf("String length [%d] Threads [%d]\n", l, THREADS);
	printf("%lu - Original calculated Blocks\n", blocks);

	int divide = (blocks + (pow(2, 31) - 1) - 1) / (pow(2, 31) - 1);
	blocks = (blocks + divide - 1) / divide;

	printf("%.0f <– Blocks limitation\n", (pow(2, 31) - 1));
	printf("%lu <- Blocks after correction\n", blocks);
	printf("%d <- Divisor\n", divide);
	printf("%lu <- Multiply check\n", divide * blocks);
	printf("%llu <- Total operations\n%lu <- Blocks per grid(s)\n", total, blocks);

	for (size_t i = 0; i < divide; i++)
	{
		for (size_t j = 0; j < 4; j++)
		{
			hipSetDevice(j);
			kernel_mult<<<blocks/4, THREADS>>>(i, blocks, j, l);
		}
		hipDeviceSynchronize();
	}
}